#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <random>
#include <cassert>
#include "matmul.cuh"

void fill_matrix_with_random(float* matrix, size_t n) {
    // setting up random generator 
    std::random_device entropy_source;
    std::mt19937_64 generator(entropy_source());
    std::uniform_real_distribution<float> dist(-1.0, 1.0);

    for (size_t i = 0; i < n * n; ++i) {
        matrix[i] = dist(generator);
    }
}

int main(int argc, char** argv) {
    if(argc != 3) {
      std::cerr<<"Incorrect Arguments\n";
      return 1;
    }
    size_t n = std::atoi(argv[1]); 
    unsigned int threads_per_block = std::atoi(argv[2]);

    //allocate mem for host
    std::vector<float> h_A(n * n);
    std::vector<float> h_B(n * n);
    std::vector<float> h_C(n * n, 0);


    fill_matrix_with_random(h_A.data(), n);
    fill_matrix_with_random(h_B.data(), n);

    //allocate mem for div
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, n * n * sizeof(float));
    hipMalloc(&d_B, n * n * sizeof(float));
    hipMalloc(&d_C, n * n * sizeof(float));

    // host->div
    hipMemcpy(d_A, h_A.data(), n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B.data(), n * n * sizeof(float), hipMemcpyHostToDevice);

    // timing events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    hipEventRecord(start);
    matmul(d_A, d_B, d_C, n, threads_per_block);
    hipEventRecord(stop);

    hipEventSynchronize(stop);

    hipMemcpy(h_C.data(), d_C, n * n * sizeof(float), hipMemcpyDeviceToHost);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout<< h_C[n * n - 1] << std::endl;

    std::cout << milliseconds << std::endl;

    //clean mem
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}

