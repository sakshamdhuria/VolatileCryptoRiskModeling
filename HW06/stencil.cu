#include "hip/hip_runtime.h"
#include "stencil.cuh"
#include <hip/hip_runtime.h>
#include <cassert>

__global__ void stencil_kernel(const float* image, const float* mask, float* output, unsigned int n, unsigned int R) {
    extern __shared__ float shared_mem[]; //shared mem
    //shared image and mask from the shared memory^^
    float* shared_image = shared_mem, *shared_mask = shared_mem+(blockDim.x + 2 * R); 

    unsigned int globalID = blockIdx.x * blockDim.x + threadIdx.x; 
    unsigned int localID = threadIdx.x; 

    // mask->sharedMask
    if (localID < 2 * R + 1) {
        shared_mask[localID] = mask[localID];
    }
    __syncthreads(); 

    // Load image elements needed for this block into shared memory
    int sumIter = globalID - R;
    if (sumIter<0 || sumIter>=n) {
        shared_image[localID] = 1.0f; 
    } else {
        shared_image[localID] = image[sumIter];
    }
    __syncthreads();

    if (globalID < n) {
        float result = 0.0f;
        for (int j = -R; j <= R; ++j) {
            result += shared_image[localID + j + R] * shared_mask[j + R];
        }
        output[globalID] = result;
    }
}


__host__ void stencil(const float* image,
                      const float* mask,
                      float* output,
                      unsigned int n,
                      unsigned int R,
                      unsigned int threads_per_block) {

    unsigned int num_blocks = (n + threads_per_block - 1) / threads_per_block;
    size_t shared_memory_size = (threads_per_block + 2 * R) * sizeof(float) + (2 * R + 1) * sizeof(float);
    stencil_kernel<<<num_blocks, threads_per_block, shared_memory_size>>>(image, mask, output, n, R);
    hipDeviceSynchronize();
}

