#include "stencil.cuh"
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <random>
#include <cassert>

int main(int argc, char** argv) {
    if (argc != 4) {
        std::cerr << "Incorrect Arguments" << std::endl;
        return 1;
    }
    //setup
    unsigned int n = std::atoi(argv[1]);
    unsigned int R = std::atoi(argv[2]);
    unsigned int threads_per_block = std::atoi(argv[3]);
    std::random_device entropy_source;
    std::mt19937_64 generator(entropy_source());
    std::uniform_real_distribution<float> dist(-1.0f, 1.0f);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    //allocate host mem
    std::vector<float> image(n);
    std::vector<float> mask(2 * R + 1);
    std::vector<float> output(n, 0.0f);

    //initialize random values
    for (unsigned int i = 0; i < n; ++i) {
        image[i] = dist(generator);
    }
    for (unsigned int i = 0; i < 2 * R + 1; ++i) {
        mask[i] = dist(generator);
    }

    //allocate dev mem
    float *d_image, *d_mask, *d_output;
    hipMalloc(&d_image, n *sizeof(float));
    hipMalloc(&d_mask, (2* R + 1) * sizeof(float));
    hipMalloc(&d_output, n * sizeof(float));
    hipMemcpy(d_image, image.data(), n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_mask, mask.data(), (2*R +1) * sizeof(float), hipMemcpyHostToDevice);


    hipEventRecord(start);
    stencil(d_image, d_mask, d_output, n, R, threads_per_block);
    hipEventRecord(stop);

    hipEventSynchronize(stop);

    hipMemcpy(output.data(), d_output, n * sizeof(float), hipMemcpyDeviceToHost);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout <<output[n - 1] << std::endl;

    std::cout << milliseconds << std::endl;

    //clear mem
    hipFree(d_image);
    hipFree(d_mask);
    hipFree(d_output);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}

