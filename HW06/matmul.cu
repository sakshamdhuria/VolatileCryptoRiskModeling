#include "hip/hip_runtime.h"
#include "matmul.cuh"
#include <cassert>
#include <cstdio>

__global__ void matmul_kernel(const float* A, const float* B, float* C, size_t n) {
    // Compute the row and column of the matrix element to be processed by this thread
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int row = idx / n; // row index in the matrix
    int col = idx % n; // column index in the matrix

    // Check if the thread index is valid (in case of extra threads beyond matrix size)
    if (row < n && col < n) {
        float sum = 0.0f;

        // Compute the dot product for the row of A and the column of B
        for (size_t k = 0; k < n; ++k) {
            sum += A[row * n + k] * B[k * n + col];
        }

        // Store the result in C
        C[row * n + col] = sum;
    }
}

// Host function to launch the kernel
void matmul(const float* A, const float* B, float* C, size_t n, unsigned int threads_per_block) {

    size_t num_blocks = (n*n + threads_per_block - 1) / threads_per_block;
    matmul_kernel<<<num_blocks, threads_per_block>>>(A, B, C, n); //call kernel
    hipDeviceSynchronize(); //sync
}

