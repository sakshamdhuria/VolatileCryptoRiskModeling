
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void computeFactorial() {
    int tid = threadIdx.x; 
    if (tid < 8) {
        int a = tid + 1; 
        int b = 1; 
        for (int i = 1; i <= a; ++i) {
            b *= i;
        }
        printf("%d!=%d\n", a, b);
    }
}

int main() {
    dim3 gridSize = 1;
    dim3 blockSize = 8;
    computeFactorial<<<gridSize, blockSize>>>();
    hipDeviceSynchronize();
    return 0;
}

