
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <sstream>
#include <cmath>
#include <hiprand/hiprand_kernel.h>
#include <chrono> 

// CUDA error checking macro
#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << std::endl; \
    return EXIT_FAILURE;}} while(0)

// Function to load the "Close" prices from a CSV file
std::vector<double> loadClosePricesFromCSV(const std::string &filename) {
    std::vector<double> closePrices;
    std::ifstream file(filename);
    if (!file.is_open()) {
        std::cerr << "Failed to open file: " << filename << std::endl;
        exit(EXIT_FAILURE);
    }

    std::string line;
    // Skip the header
    std::getline(file, line);

    // Read CSV data
    while (std::getline(file, line)) {
        std::stringstream ss(line);
        std::string value;
        int colIndex = 0;
        double closePrice;

        // Extract the "Close" column (assume it's the 5th column, index 4)
        while (std::getline(ss, value, ',')) {
            if (colIndex == 4) {
                closePrice = std::stod(value);
                closePrices.push_back(closePrice);
                break;
            }
            colIndex++;
        }
    }

    file.close();
    return closePrices;
}

// Calculate log returns
std::vector<double> calculateLogReturns(const std::vector<double>& prices) {
    std::vector<double> returns;
    for (size_t i = 1; i < prices.size(); ++i) {
        returns.push_back(std::log(prices[i] / prices[i - 1]));
    }
    return returns;
}

// Fit a simple GARCH(1,1) model
void fitGARCHModel(const std::vector<double> &returns, double &omega, double &alpha, double &beta) {
    omega = 0.01;
    alpha = 0.1;
    beta = 0.85;
}

// CUDA kernel: Initialize random number generator
__global__ void setupRNG(hiprandState *states, unsigned long seed, int numPaths) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numPaths) {
        hiprand_init(seed, idx, 0, &states[idx]);
    }
}

// CUDA kernel: Monte Carlo simulation using GARCH model
__global__ void monteCarloGARCH(
    double *prices,
    double *volatility,
    double omega, double alpha, double beta,
    hiprandState *states,
    int numPaths, int steps, double initialPrice) {
    
    int pathIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (pathIdx < numPaths) {
        hiprandState localState = states[pathIdx];
        double price = initialPrice;
        double sigma2 = volatility[pathIdx];

        for (int t = 0; t < steps; ++t) {
            double z = hiprand_normal_double(&localState);
            sigma2 = omega + alpha * sigma2 * z * z + beta * sigma2;
            double r = sqrt(sigma2) * z;
            price = price * exp(r);

            // Store simulation result
            prices[pathIdx * steps + t] = price;
        }
    }
}

int main() {

    // Load historical data and extract "Close" prices
    std::string filename = "bitcoin_data.csv";
    std::vector<double> closePrices = loadClosePricesFromCSV(filename);

    // Calculate log returns
    std::vector<double> returns = calculateLogReturns(closePrices);

    // Fit GARCH(1,1) model parameters
    double omega, alpha, beta;
    fitGARCHModel(returns, omega, alpha, beta);

    // CUDA parameters
    int numPaths = 1000;
    int steps = 60;
    double initialPrice = closePrices.back();

    // Host memory allocation
    std::vector<double> h_prices(numPaths * steps, 0.0);
    std::vector<double> h_volatility(numPaths, 0.01);

    // Device memory allocation
    double *d_prices, *d_volatility;
    hiprandState *d_states;
    CUDA_CALL(hipMalloc(&d_prices, numPaths * steps * sizeof(double)));
    CUDA_CALL(hipMalloc(&d_volatility, numPaths * sizeof(double)));
    CUDA_CALL(hipMalloc(&d_states, numPaths * sizeof(hiprandState)));

    // Copy data from host to device
    CUDA_CALL(hipMemcpy(d_volatility, h_volatility.data(), numPaths * sizeof(double), hipMemcpyHostToDevice));

    // CUDA configuration
    int threadsPerBlock = 256;
    int blocksPerGrid = (numPaths + threadsPerBlock - 1) / threadsPerBlock;

    // start time
    auto start = std::chrono::high_resolution_clock::now();

    // Initialize random number generator
    setupRNG<<<blocksPerGrid, threadsPerBlock>>>(d_states, time(0), numPaths);
    CUDA_CALL(hipDeviceSynchronize());

    // Launch kernel
    monteCarloGARCH<<<blocksPerGrid, threadsPerBlock>>>(
        d_prices, d_volatility, omega, alpha, beta, d_states,
        numPaths, steps, initialPrice);
    CUDA_CALL(hipDeviceSynchronize());

    // Copy results back to host
    CUDA_CALL(hipMemcpy(h_prices.data(), d_prices, numPaths * steps * sizeof(double), hipMemcpyDeviceToHost));

    // end time
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> duration = end - start; // 计算毫秒数
    std::cout << "Execution time: " << duration.count() << " ms" << std::endl;

    // Save results to CSV
    std::ofstream outFile("simulation_results.csv");
    outFile << "Path,";

    for (int t = 0; t < steps; ++t) {
        outFile << "Sec " << t << (t < steps - 1 ? "," : "\n");
    }

    for (int path = 0; path < numPaths; ++path) {
        outFile << "Path " << path + 1 << ",";
        for (int t = 0; t < steps; ++t) {
            outFile << h_prices[path * steps + t] << (t < steps - 1 ? "," : "\n");
        }
    }

    outFile.close();
    std::cout << "Simulation results saved to simulation_results.csv" << std::endl;

    // Free device memory
    CUDA_CALL(hipFree(d_prices));
    CUDA_CALL(hipFree(d_volatility));
    CUDA_CALL(hipFree(d_states));

    return 0;
}
